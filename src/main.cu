#include "hip/hip_runtime.h"
#include <iostream>
#include <cutf/memory.hpp>

namespace{
constexpr std::size_t N = 1 << 11;
using cuda_clock_t = unsigned long long;
__global__ void kernel(cuda_clock_t* start_clock, cuda_clock_t* end_clock){
	const auto tid = blockIdx.x;
	if(tid >= N)return;

	start_clock[tid] = clock64();
	end_clock[tid] = clock64();
}
}
int main(){
	auto d_start_clock = cutf::cuda::memory::get_device_unique_ptr<cuda_clock_t>(N);
	auto d_end_clock = cutf::cuda::memory::get_device_unique_ptr<cuda_clock_t>(N);
	auto h_start_clock = cutf::cuda::memory::get_host_unique_ptr<cuda_clock_t>(N);
	auto h_end_clock = cutf::cuda::memory::get_host_unique_ptr<cuda_clock_t>(N);

	kernel<<<N, 1>>>(d_start_clock.get(), d_end_clock.get());

	cutf::cuda::memory::copy(h_start_clock.get(), d_start_clock.get(), N);
	cutf::cuda::memory::copy(h_end_clock.get(), d_end_clock.get(), N);

	std::cout<<"tid,start,end"<<std::endl;
	for(std::size_t i = 0; i < N; i++){
		std::cout<<i<<","<<h_start_clock.get()[i]<<","<<h_end_clock.get()[i]<<std::endl;
	}
}
